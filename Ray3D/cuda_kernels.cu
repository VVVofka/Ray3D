#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <ctime>
#include ""
#include "cuda_functions.h"
#include <hip/hip_runtime_api.h>

#define GRID_SIZE 1024
#define BLOCK_SIZE 256

__constant__ int c_gridSize = GRID_SIZE;
__device__ unsigned long long* d_data;

// Вспомогательные функции
__device__ float3 normalize(float3 v){
    float length = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
    if(length > 0.0f){
        v.x /= length;
        v.y /= length;
        v.z /= length;
    }
    return v;
}

__device__ int3 wrap_position(int3 pos){
    pos.x = (pos.x + c_gridSize) % c_gridSize;
    pos.y = (pos.y + c_gridSize) % c_gridSize;
    pos.z = (pos.z + c_gridSize) % c_gridSize;
    return pos;
}

__device__ bool get_voxel(int3 pos){
    pos = wrap_position(pos);
    int linear_idx = pos.z * c_gridSize * c_gridSize + pos.y * c_gridSize + pos.x;
    int array_idx = linear_idx / 64;
    int bit_idx = linear_idx % 64;
    return (d_data[array_idx] >> bit_idx) & 1ULL;
}

// Атомарная OR операция для unsigned long long
__device__ unsigned long long atomicOrULL(unsigned long long* address, unsigned long long val){
    unsigned long long* address_as_ull = (unsigned long long*)address;
    unsigned long long old = *address_as_ull, assumed;

    do{
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, assumed | val);
    } while(assumed != old);

    return old;
}

// Альтернативная реализация записи в поверхность
__device__ void write_pixel(uchar4* output, int x, int y, int width, uchar4 color){
    if(x >= 0 && x < width && y >= 0){
        output[y * width + x] = color;
    }
}

__global__ void init_kernel(float density, hiprandState* states){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= (c_gridSize * c_gridSize * c_gridSize + 63) / 64) return;

    hiprandState localState = states[idx];
    unsigned long long random_bits = 0;

    for(int i = 0; i < 64; i++){
        float rnd = hiprand_uniform(&localState);
        if(rnd < density){
            random_bits |= (1ULL << i);
        }
    }
    d_data[idx] = random_bits;
    states[idx] = localState;
}

__global__ void update_kernel(hiprandState* states){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= (c_gridSize * c_gridSize * c_gridSize + 63) / 64) return;

    hiprandState localState = states[idx];
    unsigned long long current_bits = d_data[idx];

    // Сначала очищаем текущие биты
    d_data[idx] = 0;

    // Обрабатываем каждый установленный бит
    for(int bit = 0; bit < 64; bit++){
        if(current_bits & (1ULL << bit)){
            int linear_idx = idx * 64 + bit;
            int3 pos = {
                linear_idx % c_gridSize,
                (linear_idx / c_gridSize) % c_gridSize,
                linear_idx / (c_gridSize * c_gridSize)
            };

            // Генерируем случайное направление (0-5)
            int direction = hiprand(&localState) % 6;
            int3 new_pos = pos;

            switch(direction){
            case 0: new_pos.x = (new_pos.x + 1) % c_gridSize; break;
            case 1: new_pos.x = (new_pos.x - 1 + c_gridSize) % c_gridSize; break;
            case 2: new_pos.y = (new_pos.y + 1) % c_gridSize; break;
            case 3: new_pos.y = (new_pos.y - 1 + c_gridSize) % c_gridSize; break;
            case 4: new_pos.z = (new_pos.z + 1) % c_gridSize; break;
            case 5: new_pos.z = (new_pos.z - 1 + c_gridSize) % c_gridSize; break;
            }

            // Вычисляем новую позицию в массиве
            int new_linear_idx = new_pos.z * c_gridSize * c_gridSize +
                new_pos.y * c_gridSize + new_pos.x;
            int new_array_idx = new_linear_idx / 64;
            int new_bit_idx = new_linear_idx % 64;

            // Атомарно устанавливаем бит в новой позиции
            atomicOrULL(&d_data[new_array_idx], 1ULL << new_bit_idx);
        }
    }
    states[idx] = localState;
}

__global__ void render_kernel(hipSurfaceObject_t surface, int width, int height, float time){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width || y >= height) 
        return;

    // Вычисляем направление луча
    float aspect = (float)width / height;
    float u = (2.0f * x / width - 1.0f) * aspect;
    float v = 1.0f - 2.0f * y / height;

    // Камера с вращением
    float camAngle = time * 0.1f;
    float3 rayDir = normalize(make_float3(
        u * cosf(camAngle) - sinf(camAngle),
        v,
        -1.0f
    ));

    // Позиция камеры
    float3 cameraPos = make_float3(
        c_gridSize / 2.0f + sinf(time * 0.05f) * 100.0f,
        c_gridSize / 2.0f,
        100.0f // c_gridSize / 200000.0f
    );

    float3 rayPos = cameraPos;
    uchar4 color = make_uchar4(0, 0, 0, 255);

    // Ray marching
    for(int i = 0; i < 200; i++){
        int3 voxelPos = make_int3(
            (int)floorf(rayPos.x),
            (int)floorf(rayPos.y),
            (int)floorf(rayPos.z)
        );

        if(get_voxel(voxelPos)){
            // Жёлтый цвет
            color = make_uchar4(255, 255, 0, 255);
            break;
        }

        rayPos.x += rayDir.x * 1.5f;
        rayPos.y += rayDir.y * 1.5f;
        rayPos.z += rayDir.z * 1.5f;

        // Проверяем выход за границы
        if(rayPos.x < -100 || rayPos.x > c_gridSize + 100 ||
            rayPos.y < -100 || rayPos.y > c_gridSize + 100 ||
            rayPos.z < -10000 || rayPos.z > c_gridSize + 10000){
            break;
        }
    }
    const auto szx = x * sizeof(uchar4);
    // Записываем пиксель через surface
    surf2Dwrite(color, surface, szx, y);
}
// ***
//__global__ void render_kernel(hipSurfaceObject_t surface, int width, int height, float time){
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//    if(x >= width || y >= height) return;
//
//    // Простой тест - закрашиваем экран градиентом
//    uchar4 color;
//    if(time < 0.0f){ // Первые 2 секунды - тестовый градиент
//        color = make_uchar4(
//            (x * 255) / width,
//            (y * 255) / height,
//            128,
//            255
//        );
//    } else{
//        // Вычисляем направление луча
//        float aspect = (float)width / height;
//        float u = (2.0f * x / width - 1.0f) * aspect;
//        float v = 1.0f - 2.0f * y / height;
//
//        // Упрощенная камера - смотрим прямо в центр
//        float3 rayDir = normalize(make_float3(u, v, -1.0f));
//        float3 cameraPos = make_float3(c_gridSize / 2.0f, c_gridSize / 2.0f, c_gridSize * 1000000.0f);
//        float3 rayPos = cameraPos;
//
//        color = make_uchar4(0, 0, 0, 255);
//
//        // Ray marching с отладочной информацией
//        for(int i = 0; i < 100; i++){
//            int3 voxelPos = make_int3(
//                (int)floorf(rayPos.x),
//                (int)floorf(rayPos.y),
//                (int)floorf(rayPos.z)
//            );
//
//            // Проверяем границы
//            if(voxelPos.x < 0 || voxelPos.x >= c_gridSize ||
//                voxelPos.y < 0 || voxelPos.y >= c_gridSize //||
//                //voxelPos.z < 0 || voxelPos.z >= c_gridSize
//                ){
//                break;
//            }
//
//            if(get_voxel(voxelPos)){
//                color = make_uchar4(255, 255, 0, 255);
//                break;
//            }
//
//            rayPos.x += rayDir.x * 2.0f;
//            rayPos.y += rayDir.y * 2.0f;
//            rayPos.z += rayDir.z * 2.0f;
//        }
//    }
//
//    surf2Dwrite(color, surface, x * sizeof(uchar4), y);
//}
// ***
// Вызовите в cuda_init после инициализации
extern "C" void cuda_init(unsigned long long* data, int gridSize, float density){
    const void* p_gridSize = static_cast<const void*>(&c_gridSize); // Не удалять p_gridSize: используется для избежания красного подчёркивания в CUDACHECK
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(p_gridSize), &gridSize, sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_data), &data, sizeof(unsigned long long*)));

    size_t num_elements = (gridSize * gridSize * gridSize + 63) / 64;
    size_t state_size = num_elements * sizeof(hiprandState);
    hiprandState* states;
    checkCudaErrors(hipMalloc(&states, state_size));

    // Инициализируем генератор случайных чисел
    hiprandGenerator_t gen;
    checkCurandErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, 1234));

    // Заполняем states случайными значениями
    float* temp;
    checkCudaErrors(hipMalloc(&temp, state_size));
    checkCurandErrors(hiprandGenerateUniform(gen, temp, state_size / sizeof(float)));
    checkCudaErrors(hipMemcpy(states, temp, state_size, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipFree(temp));

    dim3 blocks(((unsigned)num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE);
    init_kernel << <blocks, BLOCK_SIZE >> > (density, states);
    checkCudaErrors(hipDeviceSynchronize());


    checkCudaErrors(hipFree(states));
    checkCurandErrors(hiprandDestroyGenerator(gen));
}

extern "C" void cuda_update(unsigned long long* data, int gridSize){
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_data), &data, sizeof(unsigned long long*)));

    size_t num_elements = (gridSize * gridSize * gridSize + 63) / 64;
    size_t state_size = num_elements * sizeof(hiprandState);
    hiprandState* states;
    checkCudaErrors(hipMalloc(&states, state_size));

    hiprandGenerator_t gen;
    checkCurandErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, clock()));

    float* temp;
    checkCudaErrors(hipMalloc(&temp, state_size));
    checkCurandErrors(hiprandGenerateUniform(gen, temp, state_size / sizeof(float)));
    checkCudaErrors(hipMemcpy(states, temp, state_size, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipFree(temp));

    dim3 blocks(((unsigned)num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE);
    update_kernel << <blocks, BLOCK_SIZE >> > (states);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipFree(states));
    checkCurandErrors(hiprandDestroyGenerator(gen));
}

extern "C" void cuda_render(hipGraphicsResource* resource, unsigned long long* data,
                           int gridSize, int width, int height){
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_data), &data, sizeof(unsigned long long*)));

    // Получаем hipArray для текстуры
    hipArray* textureArray;
    checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&textureArray, resource, 0, 0));

    // Создаем surface object для записи
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = textureArray;

    hipSurfaceObject_t surfaceObj;
    checkCudaErrors(hipCreateSurfaceObject(&surfaceObj, &resDesc));

    // Рендерим сцену
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
                 (height + blockDim.y - 1) / blockDim.y);

    static float time = 0.0f;
    time += 0.016f;

    // Используем surface для записи
    render_kernel << <gridDim, blockDim >> > (surfaceObj, width, height, time);
    //- checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Удаляем surface object
    checkCudaErrors(hipDestroySurfaceObject(surfaceObj));
}